#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>

#define BLOCKS_NUM 4
#define BLOCK_SIZE 256
#define DATA_TYPE int


__global__ void reduce( DATA_TYPE* in, DATA_TYPE* out ){
	__shared__ int data[BLOCK_SIZE];

	int tid = threadIdx.x;
	int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

	data[ tid ] = in [ i ] + in[ i + blockDim.x ];
	__syncthreads();

	for ( int s = blockDim.x / 2; s > 0; s >>= 1 ){
		if ( tid < s) data[tid] += data[tid + s];
		__syncthreads();
	}
	
	__syncthreads();

	if ( tid == 0 ) out[blockIdx.x] = data[0];
}

using namespace std;

int main(){
	DATA_TYPE nums[ BLOCKS_NUM * BLOCK_SIZE ];

	int res = 0;
	srand(time(0));
	for( int i = 0; i < BLOCKS_NUM * BLOCK_SIZE; i++ ){
		if ( i < 1000 ) nums[ i ] = rand()%100 - 50;
		else nums[ i ] = 0;
		res += nums[ i ];
	}

	cout << "For summ: " << res << endl;

	hipSetDevice( 0 );
	DATA_TYPE* in;
	DATA_TYPE* out;
	
	unsigned int in_memory_size = sizeof( DATA_TYPE ) * BLOCKS_NUM * BLOCK_SIZE;
	unsigned int out_memory_size = sizeof( DATA_TYPE ) * BLOCKS_NUM;

	hipMalloc( ( void** ) &in, in_memory_size );
	hipMalloc( ( void** ) &out, out_memory_size );
	
	hipMemcpy( in, nums, in_memory_size, hipMemcpyHostToDevice );
	
	
	dim3 block( BLOCK_SIZE );
	dim3 grid( BLOCKS_NUM );
	
	reduce<<< grid, block >>>( in, out );
	hipDeviceSynchronize();
	hipMemcpy( nums, out, out_memory_size, hipMemcpyDeviceToHost );
		
	res = 0;
	for (int i = 0; i < 2; i++) res += nums[i];
	cout << "CUDA summ: " << res << endl;

	cin.get();

	hipFree( in );
	hipFree( out );
	return 0;	
}

